#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <>
#include "hip/hip_runtime_api.h"


// Initialize a vector
void init_vector(float *a, int n)
{
    for(int i=0; i<n; i++)
    {
        a[i] = (float)(rand() % 100);
    }
}

// Verify the result
void verify_result(float *a, float *b, float *c, float factor, int n)
{
    for(int i=0; i<n; i++)
    {
        assert(c[i] == factor*a[i]+b[i]);
    }
}

int main()
{
    // Vector size
	int n = 1 << 16;
	size_t bytes = n * sizeof(float);
	
	// Declare vector pointers
	float *h_a, *h_b, *h_c;
	float *d_a, *d_b;

	// Allocate memory
	h_a = (float*)malloc(bytes);
	h_b = (float*)malloc(bytes);
	h_c = (float*)malloc(bytes);
	hipMalloc(&d_a, bytes);
	hipMalloc(&d_b, bytes);

	// Initialize vectors
	init_vector(h_a, n);
	init_vector(h_b, n);

    // Create and initialize a new context
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Copy the vector to the device
    hipblasSetVector(n, sizeof(float), h_a, 1, d_a, 1); // 1: step size
    hipblasSetVector(n, sizeof(float), h_b, 1, d_b, 1);

    // Launch simple sxapy kernel single precision a*x + y
    const float scale = 2.0f;
    hipblasSaxpy(handle, n, &scale, d_a, 1, d_b, 1);
    
    // Copy the result to host device
    hipblasGetVector(n, sizeof(float), d_b, 1, h_c, 1);

    verify_result(h_a, h_b, h_c, scale, n);

    // Clean up the vreated handle
    hipblasDestroy(handle);

    // Release allocated memory
    hipFree(d_a);
    hipFree(d_b);
    free(h_a);
    free(h_b);
    std::cout<<"Successull execuation"<<std::endl;
    return 0;
}