#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include "hip/hip_runtime_api.h"
#include "add_kernel.cuh"

__global__ void add(int* a, int* b, int* c, int n)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n)
    {
        c[idx] = a[idx] + b[idx];
    }
}

void addKernelCuda(int* a, int* b, int* c, int n)
{
    int *d_a, *d_b, *d_c; 
    size_t size = n * sizeof(int); 
    // Allocate device memory 
    checkCudaErrors(hipMalloc((void**)&d_a, size)); 
    checkCudaErrors(hipMalloc((void**)&d_b, size)); 
    checkCudaErrors(hipMalloc((void**)&d_c, size)); 
    
    // Copy inputs to device 
    checkCudaErrors(hipMemcpy(d_a, a, size, hipMemcpyHostToDevice)); 
    checkCudaErrors(hipMemcpy(d_b, b, size, hipMemcpyHostToDevice)); 
    
    // Launch kernel with 256 threads per block and n/256 blocks 
    add<<<(n + 255) / 256, 256>>>(d_a, d_b, d_c, n); 
    
    // Copy result back to host 
    checkCudaErrors(hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost)); 
    // Free device memory 
    checkCudaErrors(hipFree(d_a)); 
    checkCudaErrors(hipFree(d_b)); 
    checkCudaErrors(hipFree(d_c));
}