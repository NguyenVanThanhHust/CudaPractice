#include "hip/hip_runtime.h"
#include "template_add.cuh"

namespace SampleNamespace {

    // CUDA kernel definition
    template <typename T>
    __global__ void addKernel(const T* a, const T* b, T* c, int n) {
        int index = threadIdx.x + blockIdx.x * blockDim.x;
        if (index < n) {
            c[index] = a[index] + b[index];
        }
    }

    // Template function definition
    template <typename T>
    void add(const T* a, const T* b, T* c, int n) {
        T *d_a, *d_b, *d_c;
        size_t size = n * sizeof(T);

        // Allocate device memory
        hipMalloc((void**)&d_a, size);
        hipMalloc((void**)&d_b, size);
        hipMalloc((void**)&d_c, size);

        // Copy inputs to device
        hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

        // Launch kernel with 256 threads per block
        addKernel<<<(n + 255) / 256, 256>>>(d_a, d_b, d_c, n);

        // Copy result back to host
        hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

        // Free device memory
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
    }

    // Explicit template instantiations
    template void add<int>(const int* a, const int* b, int* c, int n);
    template void add<float>(const float* a, const float* b, float* c, int n);

} // namespace SampleNamespace
