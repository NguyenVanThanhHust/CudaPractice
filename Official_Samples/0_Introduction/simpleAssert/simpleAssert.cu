#include "hip/hip_runtime.h"
// includes, system
#ifdef _WIN32
#define WINDOWS_LEAN_AND_MEAN
#define NOMINMAX
#include <windows.h>
#else
#include <sys/utsname.h>
#endif
#include <iostream>
#include <cstring>

// includes CUDA Runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>  // helper utility functions

#ifndef MAX
#define MAX(a, b) (a>b ? a:b)
#endif

using std::cin;
using std::cout;
using std::endl;


const char *sampleName = "simpleAssert";

////////////////////////////////////////////////////////////////////////////////
// Auto-Verification Code
bool testResult = true;

////////////////////////////////////////////////////////////////////////////////
// Kernels
////////////////////////////////////////////////////////////////////////////////
//! Tests assert function.
//! Thread whose id > N will print assertion failed error message.
////////////////////////////////////////////////////////////////////////////////
__global__ void testKernel(int N) {
    int gtid = blockIdx.x * blockDim.x + threadIdx.x;
    assert(gtid < N);
}

void runTest(int argc, char **argv);

int main(int argc, char **argv) {
    printf("%s starting...\n", sampleName);

    runTest(argc, argv);

    printf("%s completed, returned %s\n", sampleName,
            testResult ? "OK" : "ERROR!");
    exit(testResult ? EXIT_SUCCESS : EXIT_FAILURE);
    }

    void runTest(int argc, char **argv) {
    int Nblocks = 2;
    int Nthreads = 32;
    hipError_t error;

    #ifndef _WIN32
    utsname OS_System_Type;
    uname(&OS_System_Type);

    printf("OS_System_Type.release = %s\n", OS_System_Type.release);

    if (!strcasecmp(OS_System_Type.sysname, "Darwin")) {
        printf("simpleAssert is not current supported on Mac OSX\n\n");
        exit(EXIT_SUCCESS);
    } else {
        printf("OS Info: <%s>\n\n", OS_System_Type.version);
    }

    #endif

    // This will pick the best possible CUDA capable device
    findCudaDevice(argc, (const char **)argv);

    // Kernel configuration, where a one-dimensional
    // grid and one-dimensional blocks are configured.
    dim3 dimGrid(Nblocks);
    dim3 dimBlock(Nthreads);

    printf("Launch kernel to generate assertion failures\n");
    testKernel<<<dimGrid, dimBlock>>>(60);

    // Synchronize (flushes assert output).
    printf("\n-- Begin assert output\n\n");
    error = hipDeviceSynchronize();
    printf("\n-- End assert output\n\n");

    // Check for errors and failed asserts in asynchronous kernel launch.
    if (error == hipErrorAssert) {
        printf(
            "Device assert failed as expected, "
            "CUDA error message is: %s\n\n",
            hipGetErrorString(error));
    }

    testResult = error == hipErrorAssert;
}