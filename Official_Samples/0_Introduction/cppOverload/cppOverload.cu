// includes, system
#include <iostream>
#include <cstring>

// includes CUDA Runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>  // helper utility functions

#ifndef MAX
#define MAX(a, b) (a>b ? a:b)
#endif

using std::cin;
using std::cout;
using std::endl;


int main(int argc, char *argv[])
{
    int devID;
    hipDeviceProp_t deviceProps;

    cout<<"Starting..."<<argv[0]<<endl;

    // Pick the best possible CUDA capable device
    devID = findCudaDevice(argc, (const char **)argv);
    
    // get GPU information
    checkCudaErrors(hipGetDevice(&devID));
    checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID));
    printf("Device %d: \"%s\" with Compute %d.%d capability\n", devID, deviceProps.name, deviceProps.major, deviceProps.minor);

    printf("printf() is called. Output: \n \n");
    // Kernel config, create 2 dim grid, each location 
    // have 3-dim block
    dim3 dimGrid(2, 2);
    dim3 dimBlock(2, 2, 2);
    testKernel<<<dimGrid, dimBlock>>>(10);
    hipDeviceSynchronize();

    return EXIT_SUCCESS;
}