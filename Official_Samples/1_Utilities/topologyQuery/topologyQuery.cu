
/*
 * This is a simple test program to measure the memcopy bandwidth of the GPU.
 * It can measure device to device copy bandwidth, host to device copy bandwidth
 * for pageable and pinned memory, and device to host copy bandwidth for
 * pageable and pinned memory.
 *
 * Usage:
 * ./bandwidthTest [option]...
 */

// CUDA runtime
#include <hip/hip_runtime.h>

// includes
#include <hip/hip_runtime_api.h>  // helper functions for CUDA error checking and initialization
#include <helper_functions.h>  // helper for shared functions common to CUDA Samples

#include <hip/hip_runtime.h>

#include <cassert>
#include <iostream>
#include <memory>

static const char *sSDKsample = "CUDA Bandwidth Test";

int main(int argc, char **argv) {
  int deviceCount = 0;
  checkCudaErrors(hipGetDeviceCount(&deviceCount));

  // Enumerates Device <-> Device links
  for (int device1 = 0; device1 < deviceCount; device1++) {
    for (int device2 = 0; device2 < deviceCount; device2++) {
      if (device1 == device2) continue;

      int perfRank = 0;
      int atomicSupported = 0;
      int accessSupported = 0;

      checkCudaErrors(hipDeviceGetP2PAttribute(
          &accessSupported, hipDevP2PAttrAccessSupported, device1, device2));
      checkCudaErrors(hipDeviceGetP2PAttribute(
          &perfRank, hipDevP2PAttrPerformanceRank, device1, device2));
      checkCudaErrors(hipDeviceGetP2PAttribute(
          &atomicSupported, hipDevP2PAttrNativeAtomicSupported, device1,
          device2));

      if (accessSupported) {
        std::cout << "GPU" << device1 << " <-> GPU" << device2 << ":"
                  << std::endl;
        std::cout << "  * Atomic Supported: "
                  << (atomicSupported ? "yes" : "no") << std::endl;
        std::cout << "  * Perf Rank: " << perfRank << std::endl;
      }
    }
  }

  // Enumerates Device <-> Host links
  for (int device = 0; device < deviceCount; device++) {
    int atomicSupported = 0;
    checkCudaErrors(hipDeviceGetAttribute(
        &atomicSupported, hipDeviceAttributeHostNativeAtomicSupported, device));
    std::cout << "GPU" << device << " <-> CPU:" << std::endl;
    std::cout << "  * Atomic Supported: " << (atomicSupported ? "yes" : "no")
              << std::endl;
  }

  return 0;
}